#include "hip/hip_runtime.h"
#include<stdio.h>
#include<conio.h>

#define WIN32_LEAN_AND_MEAN
#include <windows.h>
#include <tchar.h>
#include <stdio.h>
#include <string.h>
#include<conio.h>
#include<stdlib.h>
#include<tmmintrin.h>
#include "hip/hip_runtime.h"
#include ""
#include<hip/hip_runtime.h>

#define N 30

#pragma once
#pragma warning(disable : 4035)

int *row, *col, *ptr;
double *data;

inline unsigned __int64 GetCycleCount(void)
{
	_asm    _emit 0x0F
	_asm    _emit 0x31
}

class MyTimer
{
	unsigned __int64  m_startcycle;

public:

	unsigned __int64  m_overhead;

	MyTimer(void)
	{
		m_overhead = 0;
		Start();
		m_overhead = Stop();
	}

	void Start(void)
	{
		m_startcycle = GetCycleCount();
	}

	unsigned __int64 Stop(void)
	{
		return GetCycleCount() - m_startcycle - m_overhead;
	}
};


__global__ void csrSpmv(double *x,double * y,int * ptr,int * indices,int * data, int * limiter)
{
	int i=threadIdx.x;
	int startrow = limiter[blockIdx.x];
	int stoprow = limiter[blockIdx.x+1];
	int nnz = ptr[stoprow] - ptr[startrow];
	int numrows = stoprow - startrow;
	int t = ptr[startrow];

	__shared__ double LDS[N];
	
	if(i<nnz)
	{
		LDS[i] = data[t+i] * x[indices[t+i]];
	}
     
	if ((startrow+i) < stoprow)
	{
		double temp =0.0;
		for(int j=(ptr[startrow+i]-t);j<(ptr[startrow+i+1]-t);j++)
			temp = temp+LDS[j];
		y[startrow+i]=temp;
	}	
}

void csr(int nr, int nc, int nonz)
{
     FILE *f1=fopen("CSRsample.txt","w");
     int k;
     for(k=0;k<=nr;k++)
     {
        *(ptr+k)=0;
    }
          
     int i,j=0;
     k=0;
     for(i=1;i<=nr;i++)
     {
     	 *(ptr+i)=*(ptr+i-1);
         for(j=k;j<=nonz;j++)
         {
         	if(*(row+j)==i)
         	{
         		*(ptr+i)=*(ptr+i)+1;
         		k++;
         	}
         	else
         		break;
         }
     }
     fprintf(f1, "%d %d %d\n",nr,nc,nonz);
     for(i=0;i<nonz;i++)
     {
         fprintf(f1, "%d ",*(col+i));
         fprintf(f1, "%lf\n",*(data+i));
     }
     for(i=0;i<=nr;i++)
     {
        fprintf(f1, "%d ",*(ptr+i));
       // printf("%d",*(ptr+i));
	 }
         
     fclose(f1);
	 
	/*for(i=0;i<nonz;i++)
     {
         printf("col vlus %d\t ",*(col+i));
         printf("data values %lf\t",*(data+i));
		 printf("pointer values %d \t",*(ptr+i));
     }*/
         
 }
 
 void sort(int nz)
 {
 	int i,j,temp,posn;
 	float t;
 	for(i=0;i<nz-1;i++)
	 {
	 	posn=i;
	 	for(j=i+1;j<nz;j++)
		 {
		 	if(*(row+posn)>*(row+j))
		 	    posn=j;
		 }	
		 if(posn != i)
		 {
		 	temp=*(row+i);
		 	*(row+i)=*(row+posn);
		 	*(row+posn)=temp;
		 	
		 	temp=*(col+i);
		 	*(col+i)=*(col+posn);
		 	*(col+posn)=temp;
		 	
		 	t=*(data+i);
		 	*(data+i)=*(data+posn);
		 	*(data+posn)=t;
		 }
	 }
	
	 printf("Sorting completed");
 }
 

int main()
{
    int h;
    int w;
    double * hostInputImageData;
    double * hostOutputImageData;
    int * hostptr;
	int * hostcol;
	double * hostdata;
	int * hostdelimiters;
    double * deviceInputImageData;
    double * deviceOutputImageData;
    int * deviceptr;
	int * devicecol;
	int * devicedata;
	int * devicedelimiters;
	char line[1024];
    
  FILE *fp, *f1, *f2; 
  int i,j,nnz;
  int c=1, sum=0;

  fp = fopen("Na5.txt", "r");
  
  fgets(line, 128, fp);
  while (line[0] == '%') 
  {
    fgets(line, 128, fp); 
  }
  sscanf(line,"%d %d %d\n", &h, &w, &nnz);
  printf("%d, %d, %d\n", h, w, nnz);
  row = (int *) malloc(nnz*sizeof(int));
  col = (int *) malloc(nnz*sizeof(int));
  ptr = (int *) malloc((h+1)*sizeof(int));
  data = (double *) malloc(nnz*sizeof(double));
  hostdelimiters = (int *) malloc(h*sizeof(int));

  for(i=0;i<nnz;i++)
  {
      fscanf(fp, "%d %d %lf\n", &(row[i]),&(col[i]),&(data[i]));
  }

  fclose(fp);

  sort(nnz);
  csr(h, w, nnz);
  
  hostcol = (int *) malloc(nnz*sizeof(int));
  hostdata = (double *) malloc(nnz*sizeof(double));
  hostptr = (int *) malloc((h+1)*sizeof(int));
  hostInputImageData = (double *) malloc(w*sizeof(double));
  hostOutputImageData = (double *) malloc(w*sizeof(double));

  for(i=0;i<nnz;i++)
  {
		*(hostcol+i) = col[i];
		*(hostdata+i) = data[i];
  }
  for(i=0;i<=h;i++)
  {
		*(hostptr+i) = ptr[i];
  }
  
  for(i=0;i<w;i++)
  {
		*(hostInputImageData+i) = 1;
  }  

  hostdelimiters[0]=0;
  for(i=1;i<h;i++)
  {
	  sum += hostptr[i]-hostptr[i-1];
	  if(sum < N)
		  continue;
	  else if(sum > N)
	  {
		  hostdelimiters[c] = i-1;
		  printf("%d ",hostdelimiters[c]);
		  i--;
		  c++;
		  sum = 0;
	  }
	  else
	  {
		  hostdelimiters[c]=i;
		  sum = 0;
		  c++;
	  }
  }
  hostdelimiters[++c]=h;

    hipMalloc((void **) &deviceInputImageData, h * sizeof(int));
    hipMalloc((void **) &deviceOutputImageData, h * sizeof(int));
    hipMalloc((void **) &deviceptr, (h +1) * sizeof(int));
	hipMalloc((void **) &devicecol, nnz * sizeof(int));
	hipMalloc((void **) &devicedata, nnz * sizeof(double));
	hipMalloc((void **) &devicedelimiters, h * sizeof(int));

	hipMemcpy(deviceInputImageData, hostInputImageData, h * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceptr, hostptr, (h +1) * sizeof(int) ,hipMemcpyHostToDevice);
	hipMemcpy(devicecol, hostcol, nnz * sizeof(int) ,hipMemcpyHostToDevice);
	hipMemcpy(devicedata, hostdata, nnz * sizeof(double) ,hipMemcpyHostToDevice);
    hipMemcpy(devicedelimiters, hostdelimiters, h * sizeof(int), hipMemcpyHostToDevice);

	dim3 block(32,1,1);
	dim3 grid( (nnz)/N , 1 , 1);

	MyTimer timer;

	timer.Start();                // take initial reading
	Sleep(1000);                // wait a second 
	unsigned cpuspeed10 = (unsigned)(timer.Stop() / 100000);
	
	timer.Start();
	
	csrSpmv<<<grid, block>>>(deviceInputImageData, deviceOutputImageData, deviceptr, devicecol, devicedata, devicedelimiters);	

	unsigned cycle = (unsigned)timer.Stop();
	unsigned speed = (unsigned)(cycle / 100000);

	printf("\n\nTime = %d\n", cycle * 10000 / cpuspeed10);
    
    hipMemcpy(hostOutputImageData,deviceOutputImageData, h * sizeof(double),hipMemcpyDeviceToHost);
    

   	f2 = fopen("OutputC512.txt", "w");
  
  for(i=0;i<h;i++)
  {
	  fprintf(f2, "%lf\n", *(hostOutputImageData+i));
  }
  fclose(f2);

  hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceptr);
	hipFree(devicecol);
	hipFree(devicedata);
	hipFree(devicedelimiters);

    getch();
    return 0;
}
 
