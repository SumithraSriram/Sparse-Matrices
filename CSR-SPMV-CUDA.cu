#include "hip/hip_runtime.h"
#include<stdio.h>
#include<conio.h>

#define WIN32_LEAN_AND_MEAN
#include <windows.h>
#include <tchar.h>
#include <stdio.h>
#include <string.h>
#include<conio.h>
#include<stdlib.h>
#include<tmmintrin.h>
#include "hip/hip_runtime.h"
#include ""
#include<hip/hip_runtime.h>

#pragma once
#pragma warning(disable : 4035)

int *row, *col, *ptr;
double *data;

inline unsigned __int64 GetCycleCount(void)
{
	_asm    _emit 0x0F
	_asm    _emit 0x31
}

class MyTimer
{
	unsigned __int64  m_startcycle;

public:

	unsigned __int64  m_overhead;

	MyTimer(void)
	{
		m_overhead = 0;
		Start();
		m_overhead = Stop();
	}

	void Start(void)
	{
		m_startcycle = GetCycleCount();
	}

	unsigned __int64 Stop(void)
	{
		return GetCycleCount() - m_startcycle - m_overhead;
	}
};


__global__ void csrSpmv(double *x,double * y,int * ptr,int * indices,int * data,int num_rows)
{
	int row = blockDim.x * blockIdx.x + threadIdx.x ;
	if (row < num_rows)
	{
		int dot = 0;
		for (int j = ptr[row]; j < ptr[row+1]; j++) 
			dot += data[j] * x[indices[j]];

		y[row] = dot;
	}
}

void csr(int nr, int nc, int nonz)
{
     FILE *f1=fopen("CSRsample.txt","w");
     int k;
     for(k=0;k<=nr;k++)
     {
        *(ptr+k)=0;
    }
          
     int i,j=0;
     k=0;
     for(i=1;i<=nr;i++)
     {
     	 *(ptr+i)=*(ptr+i-1);
         for(j=k;j<=nonz;j++)
         {
         	if(*(row+j)==i)
         	{
         		*(ptr+i)=*(ptr+i)+1;
         		k++;
         	}
         	else
         		break;
         }
     }
     fprintf(f1, "%d %d %d\n",nr,nc,nonz);
     for(i=0;i<nonz;i++)
     {
         fprintf(f1, "%d ",*(col+i));
         fprintf(f1, "%lf\n",*(data+i));
     }
     for(i=0;i<=nr;i++)
     {
        fprintf(f1, "%d ",*(ptr+i));
       // printf("%d",*(ptr+i));
	 }
         
     fclose(f1);
	 
	/*for(i=0;i<nonz;i++)
     {
         printf("col vlus %d\t ",*(col+i));
         printf("data values %lf\t",*(data+i));
		 printf("pointer values %d \t",*(ptr+i));
     }*/
         
 }
 
 void sort(int nz)
 {
 	int i,j,temp,posn;
 	float t;
 	for(i=0;i<nz-1;i++)
	 {
	 	posn=i;
	 	for(j=i+1;j<nz;j++)
		 {
		 	if(*(row+posn)>*(row+j))
		 	    posn=j;
		 }	
		 if(posn != i)
		 {
		 	temp=*(row+i);
		 	*(row+i)=*(row+posn);
		 	*(row+posn)=temp;
		 	
		 	temp=*(col+i);
		 	*(col+i)=*(col+posn);
		 	*(col+posn)=temp;
		 	
		 	t=*(data+i);
		 	*(data+i)=*(data+posn);
		 	*(data+posn)=t;
		 }
	 }
	
	 printf("Sorting completed");
 }
 

int main()
{
    int h;
    int w;
    double * hostInputImageData;
    double * hostOutputImageData;
    int * hostptr;
	int * hostcol;
	double * hostdata;
    double * deviceInputImageData;
    double * deviceOutputImageData;
    int * deviceptr;
	int * devicecol;
	int * devicedata;
	char line[1024];
    
  FILE *fp, *f1, *f2; 
  int i,j,nnz;
  int c=1, sum=0;

  fp = fopen("ex12.txt", "r");
  
  fgets(line, 128, fp);
  while (line[0] == '%') 
  {
    fgets(line, 128, fp); 
  }
  sscanf(line,"%d %d %d\n", &h, &w, &nnz);
  printf("%d, %d, %d\n", h, w, nnz);
  row = (int *) malloc(nnz*sizeof(int));
  col = (int *) malloc(nnz*sizeof(int));
  ptr = (int *) malloc((h+1)*sizeof(int));
  data = (double *) malloc(nnz*sizeof(double));
  
  for(i=0;i<nnz;i++)
  {
      fscanf(fp, "%d %d %lf\n", &(row[i]),&(col[i]),&(data[i]));
  }

  fclose(fp);

  sort(nnz);
  csr(h, w, nnz);
  
  hostcol = (int *) malloc(nnz*sizeof(int));
  hostdata = (double *) malloc(nnz*sizeof(double));
  hostptr = (int *) malloc((h+1)*sizeof(int));
  hostInputImageData = (double *) malloc(w*sizeof(double));
  hostOutputImageData = (double *) malloc(w*sizeof(double));

  for(i=0;i<nnz;i++)
  {
		*(hostcol+i) = col[i];
		*(hostdata+i) = data[i];
  }
  for(i=0;i<=h;i++)
  {
		*(hostptr+i) = ptr[i];
  }
  
  for(i=0;i<w;i++)
  {
		*(hostInputImageData+i) = 1;
  }  

    hipMalloc((void **) &deviceInputImageData, h * sizeof(int));
    hipMalloc((void **) &deviceOutputImageData, h * sizeof(int));
    hipMalloc((void **) &deviceptr, (h +1) * sizeof(int));
	hipMalloc((void **) &devicecol, nnz * sizeof(int));
	hipMalloc((void **) &devicedata, nnz * sizeof(double));

	hipMemcpy(deviceInputImageData, hostInputImageData, h * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceptr, hostptr, (h +1) * sizeof(int) ,hipMemcpyHostToDevice);
	hipMemcpy(devicecol, hostcol, nnz * sizeof(int) ,hipMemcpyHostToDevice);
	hipMemcpy(devicedata, hostdata, nnz * sizeof(double) ,hipMemcpyHostToDevice);

	dim3 block(32,1,1);
	dim3 grid( h/32 , 1 , 1);

	MyTimer timer;

	timer.Start();                // take initial reading
	Sleep(1000);                // wait a second 
	unsigned cpuspeed10 = (unsigned)(timer.Stop() / 100000);
	
	timer.Start();
	
	csrSpmv<<<grid, block>>>(deviceInputImageData, deviceOutputImageData, deviceptr, devicecol, devicedata, h);	

	unsigned cycle = (unsigned)timer.Stop();
	unsigned speed = (unsigned)(cycle / 100000);

	printf("\n\nTime = %d\n", cycle * 10000 / cpuspeed10);
    
    hipMemcpy(hostOutputImageData,deviceOutputImageData, h * sizeof(double),hipMemcpyDeviceToHost);
    

   	f2 = fopen("OutputC512.txt", "w");
  
  for(i=0;i<h;i++)
  {
	  fprintf(f2, "%lf\n", *(hostOutputImageData+i));
  }
  fclose(f2);

  hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceptr);
	hipFree(devicecol);
	hipFree(devicedata);

    getch();
    return 0;
}
 
